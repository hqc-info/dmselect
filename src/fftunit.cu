#include "hip/hip_runtime.h"
#include <common.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
#include <fftunit.h>
#include <toolunit.h>
#include <fileunit.h>
#include <cmdunit.h>
#include <math.h>
#include <dmunit.h>

void real_to_complex(float *r, hipfftComplex **complx, int N)
{
    int i;
    (*complx) = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N);

    #pragma omp parallel for num_threads(omp_get_num_procs()/2)
    for (i = 0; i < N; i++)
    {
        (*complx)[i].x = r[i];
        (*complx)[i].y = 0;
    }
}

float* dmFrePower(float *dmFreData, int dataLen, int FFTLen){
    static int firstTime = 0;
    static hipfftHandle plan = 1;
    static hipfftComplex *complexData, *dComplexSamples;
    static float *tempData = NULL;
    static float *tempData2 = NULL;
    if(firstTime == 0)
    {   

        CHECK_CUFFT(hipfftPlan1d(&plan, dataLen, HIPFFT_C2C, 1));
        CHECK(hipMalloc((void **)&dComplexSamples, sizeof(hipfftComplex *)*dataLen));
        CHECK(hipMalloc((void **)&tempData, sizeof(float )*FFTLen));
        tempData2 = (float*)malloc(sizeof(float)*FFTLen);
        firstTime = 1;
    }
    real_to_complex(dmFreData, &complexData, dataLen);
    CHECK(hipMemcpy((void **)dComplexSamples, complexData, sizeof(hipfftComplex)*dataLen, hipMemcpyHostToDevice));
    CHECK_CUFFT(hipfftExecC2C(plan, dComplexSamples, dComplexSamples,HIPFFT_FORWARD));

    frePower<<<FFTLen/32, 32>>>(dComplexSamples, tempData);
    hipDeviceSynchronize();
    free(complexData);
    CHECK(hipMemcpy(tempData2, tempData, sizeof(float)*FFTLen, hipMemcpyDeviceToHost));
    return tempData2;
}

__global__ void frePower(hipfftComplex* fftData, float* power){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    hipfftComplex tempComplex=fftData[ix];
    power[ix] = tempComplex.x*tempComplex.x + tempComplex.y+tempComplex.y;

}



hipfftComplex* fftAllData(float *indata, int channel, int channelLen, int outPutLen)
{   // when GPU Mem is 3 times big than data
    // fft for all data(all data load in mem)
    // got the output data on the device
    int i = 0;
    int dataSize = channelLen*channel;
    hipfftComplex *complexData, *dComplexSamples;
    hipfftHandle plan = 0;
    
    unsigned long long dataOffset = 0;
    hipfftComplex *tempData = NULL;
    
    printf("\nStart FFT!\n");
    CHECK_CUFFT(hipfftPlan1d(&plan, channelLen, HIPFFT_C2C, 1));

    // 实数转换为复数，便于计算
    real_to_complex(indata, &complexData, dataSize);
    
    // 复制数据进入显卡
    printf("FFT:start copy data to device!\n");
    CHECK(hipMalloc((void **)&dComplexSamples, sizeof(hipfftComplex *)*dataSize));
    CHECK(hipMemcpy((void **)dComplexSamples, complexData, sizeof(hipfftComplex)*dataSize, hipMemcpyHostToDevice));

    double fftStart = seconds();
    for(i=0; i<channel; i++)
    {// fft计算，一个通道一个通道的进行计算
        CHECK_CUFFT(hipfftExecC2C(plan, dComplexSamples+dataOffset, dComplexSamples+dataOffset,HIPFFT_FORWARD));
        dataOffset += channelLen;
    }

    //FFT 数据取半
    CHECK(hipMalloc((void **)&tempData, sizeof(hipfftComplex *)*outPutLen*channel));
    dataOffset = 0;
    for(i=0;i<channel;i++){
        CHECK(hipMemcpy((tempData+dataOffset), (dComplexSamples+i*channelLen), sizeof(hipfftComplex)*outPutLen, hipMemcpyDeviceToDevice));
        dataOffset += outPutLen;
    }
    printf("FFT time cost:%.4fs\n", seconds()-fftStart);

    free(complexData);
    CHECK(hipFree(dComplexSamples));
    CHECK_CUFFT(hipfftDestroy(plan));
    return tempData;
}


hipfftComplex* fftPartOfData(float *indata, int channel, int channelLen, int outPutLen, int endMark)
{   // When system resources are limited
    // Processing part of the data each time
    // got the output data on the Host
    int i = 0;
    size_t dataSize = (size_t)channelLen*channel;
    hipfftComplex *dComplexSamples;//, *complexData;
    static hipfftHandle plan = 0;
    static int first = 0;

    hipfftReal *devicein;

    size_t dataOffset = 0;
    size_t fftOffset = 0;
    hipfftComplex *tempData = NULL;
    if(first == 0 ){
        CHECK_CUFFT(hipfftPlan1d(&plan, channelLen, HIPFFT_R2C, 1));
        first = 1;
    }
        
    CHECK(hipMalloc((void **)&devicein, sizeof(hipfftReal)*dataSize));
    CHECK(hipMalloc((void **)&dComplexSamples, sizeof(hipfftComplex *)*(outPutLen+1)*channel));
    CHECK(hipMemcpy((void **)devicein, indata, sizeof(hipfftReal)*dataSize, hipMemcpyHostToDevice)); 


    for(i=0; i<channel; i++)
    {// fft计算，一个通道一个通道的进行计算
        CHECK_CUFFT(hipfftExecR2C(plan, devicein+dataOffset, dComplexSamples+fftOffset));
        dataOffset += channelLen;
        fftOffset +=(outPutLen+1);
    }

    free(indata);

    tempData = (hipfftComplex*)malloc(sizeof(hipfftComplex *)*outPutLen*channel);
    dataOffset = 0;
    fftOffset = 0;
    for(i=0;i<channel;i++){
        CHECK(hipMemcpy((tempData+dataOffset), (dComplexSamples+fftOffset), sizeof(hipfftComplex)*outPutLen, hipMemcpyDeviceToHost));
        dataOffset += outPutLen;
        fftOffset +=(outPutLen+1);
    }

    CHECK(hipFree(devicein));
    CHECK(hipFree(dComplexSamples));
    if(endMark == 1){
        CHECK_CUFFT(hipfftDestroy(plan));
    }
    return tempData;
}


hipfftComplex* fftPartOfDataAndAbsSum(float *indata, int channel, int channelLen, int outPutLen, int endMark, float *absSum)
{   
    // abs data when the fft is processing
    int i = 0;
    size_t dataSize = (size_t)channelLen*channel;
    hipfftComplex *dComplexSamples;//, *complexData;
    static hipfftHandle plan = 0;
    static int first = 0;
    static float* cuAbsSum;
    // static hipfftComplex* cuAbsSum;
    static double absAddTime=0;  // abs 
    hipfftReal *devicein;

    size_t dataOffset = 0;
    size_t fftOffset = 0;
    hipfftComplex *tempData = NULL;
    if(first == 0 ){
        CHECK_CUFFT(hipfftPlan1d(&plan, channelLen, HIPFFT_R2C, 1));
        first = 1;
        CHECK(hipMalloc((void **)&cuAbsSum, sizeof(float)*(outPutLen+1)));
        CHECK(hipMemset(cuAbsSum, 0, sizeof(float)*(outPutLen+1)));
        // CHECK(hipMalloc((void **)&cuAbsSum, sizeof(hipfftComplex)*(outPutLen+1)));
        // CHECK(hipMemset(cuAbsSum, 0, sizeof(hipfftComplex)*(outPutLen+1)));
    }
        
    CHECK(hipMalloc((void **)&devicein, sizeof(hipfftReal)*dataSize));
    CHECK(hipMalloc((void **)&dComplexSamples, sizeof(hipfftComplex *)*(outPutLen+1)*channel));
    CHECK(hipMemcpy((void **)devicein, indata, sizeof(hipfftReal)*dataSize, hipMemcpyHostToDevice)); 


    for(i=0; i<channel; i++)
    {// fft计算，一个通道一个通道的进行计算
        CHECK_CUFFT(hipfftExecR2C(plan, devicein+dataOffset, dComplexSamples+fftOffset));
        dataOffset += channelLen;
        fftOffset +=(outPutLen+1);
    }

    free(indata);
    hipDeviceSynchronize();
    CHECK(hipFree(devicein));
    double templtime1 = seconds();
    // channelComplexSum<<<outPutLen/32, 32>>>(dComplexSamples, cuAbsSum, outPutLen+1, channel);  
    channelComplexSumAbs<<<outPutLen/32, 32>>>(dComplexSamples, cuAbsSum, outPutLen+1, channel); 
    hipDeviceSynchronize();
    absAddTime += seconds() - templtime1;

    tempData = (hipfftComplex*)malloc(sizeof(hipfftComplex *)*outPutLen*channel);
    dataOffset = 0;
    fftOffset = 0;
    for(i=0;i<channel;i++){
        CHECK(hipMemcpy((tempData+dataOffset), (dComplexSamples+fftOffset), sizeof(hipfftComplex)*outPutLen, hipMemcpyDeviceToHost));
        dataOffset += outPutLen;
        fftOffset +=(outPutLen+1);
    }

    
    CHECK(hipFree(dComplexSamples));
    if(endMark == 1){
        printf("\n\n\nsum and abs time cost: %f   \n\n\n", absAddTime);
        CHECK_CUFFT(hipfftDestroy(plan));
        CHECK(hipMemcpy((void **)absSum, cuAbsSum, sizeof(float)*outPutLen, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        CHECK(hipFree(cuAbsSum));
        // exit(0);
    }
    return tempData;
}


struct fileinfo fftProcess(struct fileinfo datainfo, struct systemSource source, struct cmds cmdData){
    /*
        this func is use for fft data save or data need to be deal with part
    */
    struct fileinfo outputinfo;
    outputinfo = datainfo;
    outputinfo.DataType = 2;
    outputinfo.FFTLen = datainfo.channelLen/2;
    char *filename = getFileName(datainfo.path);
    strcat(filename, ".fft");
    outputinfo.path = addPath(cmdData.outputPath, filename);
    

    int partMark = isProcessPartly(source,  datainfo.channelLen, datainfo.channel);
    
    outputinfo = writefile(outputinfo.path, outputinfo); 

    if(partMark == 0){ // Judge whether to process files separately
        float *tempData;
        tempData = readAllFloatData(datainfo);
        hipfftComplex *complexHostData = NULL;
        hipfftComplex *complexDeviceData = NULL;

        complexDeviceData = fftPartOfData(tempData, outputinfo.channel, datainfo.channelLen, outputinfo.FFTLen, 1);
        complexHostData = (hipfftComplex*)malloc(sizeof(hipfftComplex)*outputinfo.channel*outputinfo.FFTLen);
        CHECK(hipMemcpy(complexHostData, complexDeviceData,sizeof(hipfftComplex)*outputinfo.channel*outputinfo.FFTLen, hipMemcpyDeviceToHost));
        CHECK(hipFree(complexDeviceData));
        writeComplexData(outputinfo, complexHostData, (size_t)outputinfo.channel*outputinfo.FFTLen);
        free(tempData);
        free(complexHostData);
    }else{

        // func for part of data 
        hipfftComplex *complexHostData = NULL;
        int readChannel =fftReadChannelNum(source, datainfo.channelLen, datainfo.channel);
        int allChannel = datainfo.channel;
        int channelCount = 0;
        int loopTimes = ceil((float)allChannel/readChannel);
        float *tempData;
        
        // tempData = (float*)malloc(sizeof(float)*datainfo.channelLen*readChannel);
        // float *tempData2;
        // size_t offset = 0;
        double Start = seconds();
        int firstmark = 0;
        for(int i=0; i<loopTimes;i++)
        {   
            if((allChannel-i*readChannel) <readChannel){
                readChannel = allChannel-i*readChannel;
            }

            if(firstmark == 0)
            {
                firstmark = 1;
            }else{
                free(tempData);
            }
            tempData = readMultiChannelFloatData(datainfo, channelCount,readChannel); 
        

            channelCount = channelCount + readChannel;

            processBar(i, loopTimes);

            if(channelCount == allChannel){
                complexHostData = fftPartOfData(tempData, readChannel, datainfo.channelLen, datainfo.channelLen/2, 1);

            }else{
                complexHostData = fftPartOfData(tempData, readChannel, datainfo.channelLen, datainfo.channelLen/2, 0);

            }
            writeComplexData(outputinfo, complexHostData, (size_t)(datainfo.channelLen/2)*readChannel);
            fflush(outputinfo.file);

        }
        printf("FFT time cost:%.3fs\n", seconds()-Start);
        free(complexHostData);
        

    }
    closefile(outputinfo);
    closefile(datainfo);

    printf("FFT file save at: %s\n\n", outputinfo.path );
    return outputinfo;
    
}

float* fftAndAbsSumProcess(struct fileinfo datainfo, struct systemSource source, struct cmds cmdData, struct fileinfo *outputfile){
    /*
        this func is use for fft data save or data need to be deal with part
    */
    struct fileinfo outputinfo;
    outputinfo = datainfo;
    outputinfo.DataType = 2;
    outputinfo.FFTLen = datainfo.channelLen/2;
    char *filename = getFileName(datainfo.path);
    strcat(filename, ".fft");
    outputinfo.path = addPath(cmdData.outputPath, filename);
    
    float* absSum;
    absSum = (float*)malloc((size_t)sizeof(float)*outputinfo.FFTLen);
    memset(absSum, 0, sizeof(float)*outputinfo.FFTLen);
    
    
    outputinfo = writefile(outputinfo.path, outputinfo); 

    // func for part of data 
    hipfftComplex *complexHostData = NULL;
    int readChannel =fftReadChannelNum(source, datainfo.channelLen, datainfo.channel);
    int allChannel = datainfo.channel;
    int channelCount = 0;
    int loopTimes = ceil((float)allChannel/readChannel);
    float *tempData;
    
    // tempData = (float*)malloc(sizeof(float)*datainfo.channelLen*readChannel);
    // float *tempData2;
    // size_t offset = 0;
    double Start = seconds();
    int firstmark = 0;
    printf("\nFFT and ABS sum processing... could cost a little bit time\n");
    for(int i=0; i<loopTimes;i++)
    {   
        if((allChannel-i*readChannel) <readChannel){
            readChannel = allChannel-i*readChannel;
        }

        if(firstmark == 0)
        {
            firstmark = 1;
        }else{
            free(tempData);
        }
        tempData = readMultiChannelFloatData(datainfo, channelCount,readChannel); 
    

        channelCount = channelCount + readChannel;

        processBar(i, loopTimes);

        if(channelCount == allChannel){
            complexHostData = fftPartOfDataAndAbsSum(tempData, readChannel, datainfo.channelLen, datainfo.channelLen/2, 1, absSum);

        }else{
            complexHostData = fftPartOfDataAndAbsSum(tempData, readChannel, datainfo.channelLen, datainfo.channelLen/2, 0, absSum);

        }
        writeComplexData(outputinfo, complexHostData, (size_t)(datainfo.channelLen/2)*readChannel);
        fflush(outputinfo.file);

    }
    printf("FFT time cost:%.3fs\n", seconds()-Start);
    free(complexHostData);
    

    closefile(outputinfo);
    closefile(datainfo);

    printf("FFT file save at: %s\n\n", outputinfo.path );
    // copy info data to output file info
    memcpy(outputfile, &outputinfo, sizeof(struct fileinfo));
    return absSum;
    
}