#include "hip/hip_runtime.h"
#include<dmunit.h>
#include<hip/hip_runtime.h>
#include <stdio.h>
#include <common.h>
#include <hip/hip_runtime.h>

__global__ void channelComplexSumAbs(hipfftComplex *indata, float *outdata, unsigned int datalen, int channel){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int i = 0;
    hipfftComplex temp;
    float outValue=0;
    for(i=0;i<channel;i++){
        temp = indata[ix+i*datalen];
        outValue += sqrtf(temp.x*temp.x+temp.y*temp.y);
    }
    outdata[ix] += outValue;
}

__global__ void printinfotest(hipfftComplex *indata){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    printf("complex:%f\n", indata[ix].x);
}


__global__ void printinfotestf(float *indata){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    printf("float:%f\n", indata[ix]);
}


__global__ void channelComplexSum(hipfftComplex *indata, hipfftComplex *outdata, unsigned int datalen, int channel){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int i = 0;
    hipfftComplex temp;
    hipfftComplex outValue;
    for(i=0;i<channel;i++){
        outValue.x += indata[ix+i*datalen].x;
        outValue.y += indata[ix+i*datalen].y;
    }
    outdata[ix].x += outValue.x;
    outdata[ix].y += outValue.y;
}


__global__ void  calculatePicRowData(hipfftComplex* complexDmData, float* picRowData, int sumChannelNum, int channelStep, int dmNum, int channel){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int x = (ix%dmNum);
    unsigned int y = ix/dmNum;
    hipfftComplex temp;
    temp.x=0;
    temp.y=0;
    int index = 0;
    for(int loop=0; loop < sumChannelNum; loop++)
    {   
        index = (y*channelStep+loop)+x*channel;
        temp.x += complexDmData[index].x;
        temp.y += complexDmData[index].y;
    }
    picRowData[ix] = sqrtf(temp.x*temp.x+temp.y*temp.y);

}



// 筛选dm算法
__global__ void partSelectFunc(hipfftComplex *fftData, unsigned int dataIndex, unsigned int fftIndex, int *dmdata, hipfftComplex *outdata,int channelNum, unsigned int channelLen, unsigned int dataLen, int downsample){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * channelNum + ix;
    float pi2 = 6.28318534;
    hipfftComplex temp_complex;
    hipfftComplex temp_complex2;
    float s, c;
    temp_complex2 = fftData[dataIndex*channelNum+ ix];
    sincosf(pi2*dmdata[idx]*(fftIndex)/channelLen/downsample, &s, &c);
    temp_complex.x = temp_complex2.x*c + (-1)*temp_complex2.y*s;
    temp_complex.y = temp_complex2.x*s + temp_complex2.y*c;
    outdata[idx] = temp_complex;
}



// 筛选dm算法
__global__ void selectFunc(hipfftComplex *fftData, unsigned int dataIndex, int *dmdata, hipfftComplex *outdata, int channelNum, unsigned int channelLen, unsigned int dataLen, int downsample){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * channelNum + ix;
    float pi2 = 6.28318534;
    hipfftComplex temp_complex;
    hipfftComplex temp_complex2;
    float s, c;
    temp_complex2 = fftData[dataIndex+ ix*dataLen];
    sincosf(pi2*dmdata[idx]*(dataIndex)/channelLen/downsample, &s, &c);
    temp_complex.x = temp_complex2.x*c + (-1)*temp_complex2.y*s;
    temp_complex.y = temp_complex2.x*s + temp_complex2.y*c;
    outdata[idx] = temp_complex;
}


// 筛选dm值后每个通道加和
__global__ void selectSumDM(hipfftComplex *indata, hipfftComplex *outdata, int channelNum){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int i = 0;
    hipfftComplex temp_complex;
    temp_complex.x = 0;
    temp_complex.y = 0;
  
    for(i=0;i<channelNum;i++){
        temp_complex.x += indata[ix*channelNum+i].x;
        temp_complex.y += indata[ix*channelNum+i].y;
    }
    outdata[ix] = temp_complex;
  
  }

  // 筛选 abs
__global__ void selectAbsDm(hipfftComplex *indata, float *outdata){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    hipfftComplex temp = indata[ix];
    outdata[ix] = sqrtf(temp.x*temp.x+temp.y*temp.y);
  }
  